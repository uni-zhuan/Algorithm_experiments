#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <omp.h>

#define MATRIX_SIZE 1000
#define BLOCK_SIZE 16
int DevicedChoosed = 0;

void printDeviceProp(const hipDeviceProp_t& prop)
//��ӡ�豸��Ϣ
{
	printf("Device Name : %s.\n", prop.name);
	printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
	printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
	printf("regsPerBlock : %d.\n", prop.regsPerBlock);
	printf("warpSize : %d.\n", prop.warpSize);
	printf("memPitch : %d.\n", prop.memPitch);
	printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("totalConstMem : %d.\n", prop.totalConstMem);
	printf("major.minor : %d.%d.\n", prop.major, prop.minor);
	printf("clockRate : %d.\n", prop.clockRate);
	printf("textureAlignment : %d.\n", prop.textureAlignment);
	printf("deviceOverlap : %d.\n", prop.deviceOverlap);
	printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

//CUDA ��ʼ��
bool InitCUDA()
{
	int count;
	//ȡ��֧��Cuda��װ�õ���Ŀ
	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}
	int i;
	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		//��ӡ�豸��Ϣ
		printDeviceProp(prop);
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}

	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);
	DevicedChoosed = i;
	return true;
}

void matMultCPU(const float* a, const float* b, float* c, int n)
//openmp���д���
{
	omp_set_num_threads(32);
	printf("Procs: %d, Thread: %d \n", omp_get_num_procs(), omp_get_max_threads());
#pragma omp parallel for
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			double t = 0;
			for (int k = 0; k < n; k++)
			{
				t += (double)a[i * n + k] * b[k * n + j];
			}
			c[i * n + j] = t;
		}
	}
}

//GPU���м������˷�
__global__ void matMultCUDAKernel1(const float* a, const float* b, float* c, int n)
{
	//������� thread Ӧ�ü���� row �� col
	const int col = blockIdx.x * blockDim.x + threadIdx.x;
	const int row = blockIdx.y * blockDim.y + threadIdx.y;

	int i;
	//�������˷� Kahan��s Summation Formula
	if (row < n && col < n)
	{
		float t = 0;
		float y = 0;
		for (i = 0; i < n; i++)
		{
			float r;

			y -= a[row * n + i] * b[i * n + col];
			r = t - y;
			y = (r - t) + y;
			t = r;
		}
		c[row * n + col] = t;
	}
}

void genMat(float* arr, int n)
//���ɾ���
{
	int i, j;

	for (i = 0; i < n; i++)
	{
		for (j = 0; j < n; j++)
		{
			arr[i * n + j] = (float)rand() / RAND_MAX + (float)rand() / (RAND_MAX * RAND_MAX);
		}
	}

}

typedef struct Error {
	float max;
	float average;
}Error;

Error accuracyCheck(const float* a, const float* b, int n)
{
	Error err;
	err.max = 0;
	err.average = 0;
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			if (b[i * n + j] != 0)
			{
				//fabs�󸡵���x�ľ���ֵ
				float delta = fabs((a[i * n + j] - b[i * n + j]) / b[i * n + j]);
				if (err.max < delta) err.max = delta;
				err.average += delta;
			}
		}
	}
	err.average = err.average / (n * n);
	return err;
}

static void matMultCPU_serial(const float* a, const float* b, float* c, int n)
//���м���ʱ��
{
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			double t = 0;
			for (int k = 0; k < n; k++)
			{
				t += (double)a[i * n + k] * b[k * n + j];
			}
			c[i * n + j] = t;
		}
	}
}

int main(int argc, char** argv)
{
	//CUDA ��ʼ��
	if (!InitCUDA()) return 0;
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, DevicedChoosed);
	//�������
	float* a, * b, * c, * d;
	int n = MATRIX_SIZE;
	if (argc >= 2) n = atoi(argv[1]) > 0 ? atoi(argv[1]) : MATRIX_SIZE;

	//����host�ڴ�
	hipHostMalloc((void**)&a, sizeof(float) * n * n);
	hipHostMalloc((void**)&b, sizeof(float) * n * n);
	hipHostMalloc((void**)&c, sizeof(float) * n * n);
	d = (float*)malloc(sizeof(float) * n * n);

	genMat(a, n);
	genMat(b, n);

	float* cuda_a, * cuda_b, * cuda_c;
	clock_t start, stop;
	//����GPU�ϵ��ڴ�
	hipMalloc((void**)&cuda_a, sizeof(float) * n * n);
	hipMalloc((void**)&cuda_b, sizeof(float) * n * n);
	hipMalloc((void**)&cuda_c, sizeof(float) * n * n);

	//����������GPU�ڴ�
	hipMemcpy(cuda_a, a, sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(cuda_b, b, sizeof(float) * n * n, hipMemcpyHostToDevice);
	start = clock();
	//���ú˺�������
	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 gridSize((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);//����ȡ��
	matMultCUDAKernel1 <<<gridSize, blockSize >>> (cuda_a, cuda_b, cuda_c, n);

	//���������ƻ����棬��ʽ����ͬ������
	hipMemcpy(c, cuda_c, sizeof(float) * n * n, hipMemcpyDeviceToHost);
	stop = clock();
	//�ͷ�GPU�ϵ��ڴ�
	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_c);
	//GPU memory
	printf("GPU memory: %e MB\n", (double)(n * n * 8) / (1024. * 1024.));
	//GPU time
	double gpu_t = ((double)stop - start) / CLOCKS_PER_SEC * 1000.0;
	printf("GPU time: %3f ms\n", gpu_t);
	//CPU time
	//start = clock();
	////matMultCPU(a, b, d, n);
	//matMultCPU_serial(a, b, c, n);
	//stop = clock();
	//double cpu_t = ((double)stop - start) / CLOCKS_PER_SEC * 1000.0;
	//printf("CPU_serial time: %3f ms\n", cpu_t);
	//printf("CUDA S0: %3f\n", cpu_t / gpu_t);

	//start = clock();
	//matMultCPU(a, b, d, n);
	//stop = clock();
	//double cpu2_t = ((double)stop - start) / CLOCKS_PER_SEC * 1000.0;
	//printf("CPU_parallel time: %3f ms\n", cpu2_t);
	//printf("OpenMP S0: %3f\n", cpu_t / cpu2_t);

	return 0;
}